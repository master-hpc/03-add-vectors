// 
// Il est demandéd'expliquer l'ensemble des instructions
// pr�c�d�es par un TODO: ?
//


#include <hip/hip_runtime.h>
#include<iostream>

__global__ void addKernel(int *c, const int *a, const int *b);

int main(int argc, char** argv)
{
	const int size = 5;
	// TODO: ?
	const int h_a[size] = { 1, 2, 3, 4, 5 };
	const int h_b[size] = { 10, 20, 30, 40, 50 };
	
	int h_c[size] = { 0 };
	
	// TODO: ?
	hipSetDevice(0);
	
	// TODO: ?
	int *d_a = 0;
	int *d_b = 0;
	int *d_c = 0;
	
	// TODO: ?
	hipMalloc((void**)&d_a, size * sizeof(int));
	hipMalloc((void**)&d_b, size * sizeof(int));
	hipMalloc((void**)&d_c, size * sizeof(int));
	
	// TODO: ?
	hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);
	
	// TODO: ?
	dim3 grdDim = dim3(1,1,1);
	dim3 blkDim = dim3(size,1,1);
	// TODO: ?
	addKernel <<<grdDim, blkDim >>>(d_c, d_a, d_b);
	
	// TODO: ?
	hipDeviceSynchronize();
	
	// TODO: ?
	hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
	
	// TODO: ?
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	// Impression des r�sultats
	std::cout << "{1,2,3,4,5} + {10,20,30,40,50} = {" << h_c[0] << "," << h_c[1] << "," << h_c[2] << "," << h_c[3] << "," << h_c[4] << "}" << std::endl;

	return 0;
}

// TODO: ?
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // TODO: ?
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

